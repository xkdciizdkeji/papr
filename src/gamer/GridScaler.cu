
#include <hip/hip_runtime.h>
#ifdef ENABLE_CUDA
#include "GridScaler.cuh"

// --------------------------
// cuda kernel
// --------------------------

__global__ static void calculateCoarseCost(realT *coarseWireCost, realT *coarseViaCost, const realT *wireCost, const realT *viaCost,
                                           int coarseOffsetX, int coarseOffsetY, int coarseLengthX, int coarseLengthY,
                                           int coarseN, int coarseX, int coarseY, int scaleX, int scaleY, int DIRECTION, int N, int X, int Y, int LAYER)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  if (x >= coarseLengthX || y >= coarseLengthY || z >= LAYER)
    return;
  x += coarseOffsetX;
  y += coarseOffsetY;
  int idx = xyzToIdx(x, y, z, DIRECTION, coarseN);

  // wire cost
  realT wc = 0.0f;
  const realT *val = nullptr;
  int outerScale = 0;
  int outerBound = 0;
  int innerScale = 0;
  int innerBound = 0;
  if ((z & 1) ^ DIRECTION)
  {
    val = x > 0 && z > 0 ? wireCost + z * N * N + (y * scaleY) * N + (x - 1) * scaleX : nullptr;
    outerScale = scaleY;
    outerBound = min(scaleY, Y - y * scaleY);
    innerScale = scaleX;
    innerBound = min(2 * scaleX, X - (x - 1) * scaleX);
  }
  else
  {
    val = y > 0 && z > 0 ? wireCost + z * N * N + (x * scaleX) * N + (y - 1) * scaleY : nullptr;
    outerScale = scaleX;
    outerBound = min(scaleX, X - x * scaleX);
    innerScale = scaleY;
    innerBound = min(2 * scaleY, Y - (y - 1) * scaleY);
  }
  if (val)
  {
    for (int r = 0; r < outerBound; r++)
    {
      val += r * N;
      realT acc = 0.f, res = 0.f, cnt = 0.f;
      for (int t = innerScale; t < innerBound; t++)
        acc += val[t];
      for (int t = innerBound - 1, ascend = 0; t >= innerScale; t--, ascend ^= 1)
      {
        res += acc;
        cnt += 1.0f;
        if (ascend)
        {
          for (int tt = 1; tt < innerScale; tt++)
          {
            acc -= val[tt];
            res += acc;
            cnt += 1.0f;
          }
        }
        else
        {
          for (int tt = innerScale - 1; tt > 0; tt--)
          {
            acc += val[tt];
            res += acc;
            cnt += 1.0f;
          }
        }
        acc -= val[x];
      }
      wc += res / cnt;
    }
    wc /= static_cast<realT>(outerBound);
  }
  coarseWireCost[idx] = wc;

  // via cost
  realT vc = 0.f;
  for (int xx = x * scaleX, xxe = min((x + 1) * scaleX, X); xx < xxe; xx++)
    for (int yy = y * scaleY, yye = min((y + 1) * scaleY, Y); yy < yye; yy++)
      vc += viaCost[xyzToIdx(xx, yy, z, DIRECTION, N)];
  coarseViaCost[idx] = vc / static_cast<realT>(min(scaleX, X - x * scaleX) * min(scaleY, Y - y * scaleY));
}

// --------------------------------
// GridScaler
// --------------------------------
GridScaler::GridScaler(int DIRECTION, int N, int X, int Y, int LAYER, int scaleX, int scaleY)
    : DIRECTION(DIRECTION), N(N), X(X), Y(Y), LAYER(LAYER), scaleX(scaleX), scaleY(scaleY)
{
  coarseX = (X + scaleX - 1) / scaleX;
  coarseY = (Y + scaleY - 1) / scaleY;
  coarseN = std::max(coarseX, coarseY);

  devCoarseWireCost = cuda_make_shared<realT[]>(LAYER * coarseN * coarseN);
  devCoarseViaCost = cuda_make_shared<realT[]>(LAYER * coarseN * coarseN);
}

std::vector<int> GridScaler::calculateCoarsePinIndices(const std::vector<int> &pinIndices)
{
  std::vector<int> coarsePinIndices(pinIndices.size());
  std::transform(pinIndices.begin(), pinIndices.end(), coarsePinIndices.begin(), [&](int idx)
                 {
    auto [x, y, z] = idxToXYZ(idx, DIRECTION, N);
    return xyzToIdx(x / scaleX, y / scaleY, z, DIRECTION, coarseN); });
  return std::move(coarsePinIndices);
}

utils::BoxT<int> GridScaler::calculateCoarseBoudingBox(const utils::BoxT<int> &box)
{
  return utils::BoxT<int>(box.lx() / scaleX, box.ly() / scaleY, (box.hx() + scaleX - 1) / scaleY, (box.hy() + scaleY - 1) / scaleY);
}

void GridScaler::scale()
{
  scale(utils::BoxT<int>(0, 0, coarseX, coarseY));
}

void GridScaler::scale(const utils::BoxT<int> &coarseBox)
{
  int coarseOffsetX = coarseBox.lx();
  int coarseOffsetY = coarseBox.ly();
  int coarseLengthX = coarseBox.width();
  int coarseLengthY = coarseBox.height();
  calculateCoarseCost<<<dim3((coarseX + 31) / 32, (coarseY + 31) / 32, LAYER), dim3(32, 32, 1)>>>(
      devCoarseWireCost.get(), devCoarseViaCost.get(), devWireCost.get(), devViaCost.get(),
      coarseOffsetX, coarseOffsetY, coarseLengthX, coarseLengthY,
      coarseN, coarseX, coarseY, scaleX, scaleY, DIRECTION, N, X, Y, LAYER);
  checkCudaErrors(cudaDeviceSynchronize());
}
#endif